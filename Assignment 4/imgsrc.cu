
#include <hip/hip_runtime.h>
#include <iostream>
#include <bits/stdc++.h>
using namespace std;

// float calcAvg(int m,int n, vector<vector<tuple<int,int,int>>> v, int x,int y){
//     int sum=0;
//     for(int i=0;i<m;i++){
//         for (int j=0;j<n;j++){
//             int R=get<0>(v[x+i][y+j]);
//             int G=get<1>(v[x+i][y+j]);
//             int B=get<2>(v[x+i][y+j]);
//             sum+=(R+G+B);
//         }
//     }
//     float avg=sum/((float) (m*n*3));
//     return avg;
// }

__host__ void calcAvg(int m,int n, int* v, int x,int y, float* avg, int N){
    int sum=0;
    for(int i=0;i<m;i++){
        for (int j=0;j<n;j++){
            int R=v[(x+i)*3*N+(y+j)*3];
            int G=v[(x+i)*3*N+(y+j)*3+1];
            int B=v[(x+i)*3*N+(y+j)*3+2];
            sum+=(R+G+B);
            //cout<<R<<","<<G<<","<<B<<endl;
        }
    }
    *avg=sum/((float) (m*n*3));
}


pair<int,int> rotate(int x,int y, float angle){
   double const PI = 3.14159265358979323;
   float radians = angle * (PI / 180.0f);   // convert degrees to radians
   int nx = x * cos(radians) - y * sin(radians); 
   int ny = x * sin(radians) + y * cos(radians);
   return make_pair(nx,ny);
}

void rotateimg(int m,int n, float angle, float * v){
    for(int i=0;i<m;i++){
        for (int j=0;j<n;j++){
            pair<int,int> p = rotate(i,j,angle);
            v[i*2*n+j*2]=p.first;
            v[i*2*n+j*2+1]=p.second;
            //cout<<i<<","<<j<<endl;
        }
    }
}

__global__ void query(int M,int N, int m, int n, float avgq, float* poso, float* posp, float* posn, int* img, int* imgq, float* soln, float th1, float th2){
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    int i = index/N;
    int j = index% N;
    float avgo=0.0;
    float avgp=0.0;
    float avgn=0.0;
    bool valid = false;
    //0 degree
    if (!(i+poso[2*n*(m)-2]>M-1 || j+poso[2*n*(m)-1]>N-1)){
         int sum=0;
         int xmin=i;
         int ymin=j;
         int xmax=i+m;
         int ymax=j+n;
         for(int mi=xmin;mi<xmax;mi++){
             for (int ni=ymin;ni<ymax;ni++){
                int R=img[(M-1-mi)*3*N+(ni)*3];
                int G=img[(M-1-mi)*3*N+(ni)*3+1];
                int B=img[(M-1-mi)*3*N+(ni)*3+2];
                sum+=(R+G+B);
                //cout<<R<<","<<G<<","<<B<<endl;
            }
         }
        avgo=sum/((float) (m*n*3));
        if(abs(avgo-avgq)<=th2){
            long sum=0;
            for(int mi=0;mi<m;mi++){
                for(int ni=0;ni<n;ni++){
                    int R=abs(img[(M-1-i-mi)*3*N+(j+ni)*3]-imgq[(m-1-mi)*n*3+ni*3]);
                    int G=abs(img[(M-1-i-mi)*3*N+(j+ni)*3+1]-imgq[(m-1-mi)*n*3+ni*3+1]);
                    int B=abs(img[(M-1-i-mi)*3*N+(j+ni)*3+2]-imgq[(m-1-mi)*n*3+ni*3+2]);
                    sum=sum+R*R+G*G+B*B;
                }
            }
            float rmsg=sqrt(sum/((float)m*n*3));
            if (rmsg<=th1){
                valid=true;
                soln[3*index]=rmsg;
                soln[3*index+1]=avgo;
                soln[3*index+2]=1;
                //printf("%d,%d,%f,%d\n",i,j,rmsg,index);
            }
        }
    }

    // //45 degree

    // if (!( i+posp[2*(n-1)] <0 || j+posp[2*(n-1)+1]>N-1 || i+posp[2*n*m-2]>M-1 || j+posp[2*n*m-1]>N-1 || i+posp[2*n*(m-1)]>M-1 || j+posp[2*n*(m-1)+1]<0)){
    //     int sum=0;
    //     int xmin=i;
    //     int ymin=(int) j-(m/sqrt(2.0));
    //     int xmax=(int) i+((m+n)/sqrt(2.0))+1;
    //     int ymax=(int) j+(n/sqrt(2.0))+1;
    //     if(i==985 && j==239)
    //     printf("%d,%d,%d,%d\n",xmin,xmax,ymin,ymax);
    //     for(int mi=xmin;mi<xmax;mi++){
    //          for (int ni=ymin;ni<ymax;ni++){
    //             int R=img[(M-1-mi)*3*N+(ni)*3];
    //             int G=img[(M-1-mi)*3*N+(ni)*3+1];
    //             int B=img[(M-1-mi)*3*N+(ni)*3+2];
    //             sum+=(R+G+B);
    //             //cout<<R<<","<<G<<","<<B<<endl;
    //         }
    //     }
    //     avgp=sum/((float) ((xmax-xmin+1)*(ymax-ymin+1)*3));
    //     if(i==985 && j==239)
    //     printf("%d,%d,%f\n",i,j,avgp);
    //     if(abs(avgp-avgq)<th2){
    //         long sum=0;
    //         for(int mi=0;mi<m;mi++){
    //             for(int ni=0;ni<n;ni++){
    //                 int x=(int) i+posp[mi*n*2+ni*2];
    //                 int y=(int) j+posp[mi*n*2+ni*2+1];
    //                 float R=0;
    //                 float G=0;
    //                 float B=0;
    //                 for(int k=0;k<3;k++){
    //                     int x_=x+k/2;
    //                     int y_=y+k%2;
    //                     R+=(img[(M-1-x_)*3*N+3*y_]*sqrt(pow(x_-i-posp[mi*n*2+ni*2],2)+pow(y_-j-posp[mi*n*2+ni*2+1],2)));
    //                     G+=(img[(M-1-x_)*3*N+3*y_+1]*sqrt(pow(x_-i-posp[mi*n*2+ni*2],2)+pow(y_-j-posp[mi*n*2+ni*2+1],2)));
    //                     B+=(img[(M-1-x_)*3*N+3*y_+2]*sqrt(pow(x_-i-posp[mi*n*2+ni*2],2)+pow(y_-j-posp[mi*n*2+ni*2+1],2)));
    //                 }
    //                 R=abs(R-imgq[(m-1-mi)*n*3+ni*3]);
    //                 G=abs(G-imgq[(m-1-mi)*n*3+ni*3+1]);
    //                 B=abs(B-imgq[(m-1-mi)*n*3+ni*3+2]);
    //                 sum=sum+R*R+G*G+B*B;
    //             }
    //         }
    //         float rmsg=sqrt(sum/((float)m*n*3));
    //         if (rmsg<th1 && (!valid || rmsg<soln[3*index])){
    //             valid=true;
    //             soln[index]=rmsg;
    //             printf("%d,%d,%f\n",i,j,rmsg);
    //         }
    //     }
    // }

    // //-45 degree
    // if (!( i+posn[2*(n-1)] <0 || j+posn[2*(n-1)+1]>N-1 || i+posn[2*n*m-2]>M-1 || j+posn[2*n*m-1]>N-1 || i+posn[2*n*(m-1)]>M-1 || j+posn[2*n*(m-1)+1]<0)){
        
    //     int sum=0;
    //     int xmin=(int) i-(n/sqrt(2.0));
    //     int ymin=(int) j;
    //     int xmax=(int) i+(m/sqrt(2.0))+1;
    //     int ymax=(int) j+((m+n)/sqrt(2.0))+1;
    //     for(int mi=xmin;mi<xmax;mi++){
    //          for (int ni=ymin;ni<ymax;ni++){
    //             int R=img[(M-1-mi)*3*N+(ni)*3];
    //             int G=img[(M-1-mi)*3*N+(ni)*3+1];
    //             int B=img[(M-1-mi)*3*N+(ni)*3+2];
    //             sum+=(R+G+B);
    //             //cout<<R<<","<<G<<","<<B<<endl;
    //         }
    //     }
    //     avgn=sum/((float) (m*n*3));
    //     if(abs(avgn-avgq)<th2){
    //         long sum=0;
    //         for(int mi=0;mi<m;mi++){
    //             for(int ni=0;ni<n;ni++){
    //                 int x=(int) i+posn[mi*n*2+ni*2];
    //                 int y=(int) j+posn[mi*n*2+ni*2+1];
    //                 float R=0;
    //                 float G=0;
    //                 float B=0;
    //                 for(int k=0;k<3;k++){
    //                     int x_=x+k/2;
    //                     int y_=y+k%2;
    //                     R+=(img[x_*3*N+3*y_]*sqrt(pow(x_-i+posn[mi*n*2+ni*2],2)+pow(y_-j+posn[mi*n*2+ni*2+1],2)));
    //                     G+=(img[x_*3*N+3*y_+1]*sqrt(pow(x_-i+posn[mi*n*2+ni*2],2)+pow(y_-j+posn[mi*n*2+ni*2+1],2)));
    //                     B+=(img[x_*3*N+3*y_+2]*sqrt(pow(x_-i+posn[mi*n*2+ni*2],2)+pow(y_-j+posn[mi*n*2+ni*2+1],2)));
    //                 }
    //                 R=abs(R-imgq[mi*n+ni]);
    //                 G=abs(G-imgq[mi*n+ni+1]);
    //                 B=abs(B-imgq[mi*n+ni+2]);
    //                 sum=sum+R*R+G*G+B*B;
    //             }
    //         }
    //         float rmsg=sqrt(sum/((float)m*n*3));
    //         if (rmsg<th1 && (!valid || rmsg<soln[index])){
    //             valid=true;
    //             soln[index]=rmsg;
    //         }
    //     }
    // }

    if(!valid){
        soln[3*index]=-1.0;
        soln[3*index+1]=-1.0;
        soln[3*index+2]=-1.0;
    }
}

int main(int argc, char* argv[]){
    string infile=argv[1];
    string infile_q=argv[2];
    string outfile="output.txt";
    float th1=stof(argv[3]);
    float th2=stof(argv[4]);
    int n_=stoi(argv[5]);
    ifstream inf;
    inf.open(infile.c_str(),ios::in);
    string word;
    inf>>word; 
    // cout<<word;
    int M = stoi(word);
    inf>>word;
    //cout<<word;
    int N=stoi(word);
    cout<<M<<","<<N<<endl;
    // vector<vector<tuple<int,int,int>>> img;
    int * img = new int[M*N*3];
    cout<<M*N*3<<endl;
    // for(int i=0;i<M;i++){
    //     vector<tuple<int,int,int>> v(N,make_tuple(0,0,0));
    //     img.push_back(v);
    // }
    cout<<"reading started"<<endl;
    for(int i=0;i<M;i++){
        for (int j=0;j<N;j++){
            inf>>word;
            //cout<<word<<",";
            int R= stoi(word);
            img[3*N*i+3*j]=R;
            inf>>word;
            //cout<<word<<",";
            int G= stoi(word);
            img[3*N*i+3*j+1]=G;
            inf>>word;
            //cout<<word<<endl;
            int B= stoi(word);
            img[3*N*i+3*j+2]=B;
            // img[i][j]=(make_tuple(R,G,B));
            // cout<<i<<","<<j<<endl;
        }
    }
    inf.close();
    cout<<"reading done"<<endl;
    ifstream infq;
    infq.open(infile_q.c_str(),ios::in);
    infq>>word; 
    int m = stoi(word);
    infq>>word;
    int n=stoi(word);
    cout<<m<<","<<n<<endl;
    // vector<vector<tuple<int,int,int>>> imgq;
    int * imgq = new int[m*n*3];
    // for(int i=0;i<m;i++){
    //     vector<tuple<int,int,int>> v(n,make_tuple(0,0,0));
    //     imgq.push_back(v);
    // }
    cout<<"reading started"<<endl;
    for(int i=0;i<m;i++){
        for (int j=0;j<n;j++){
            infq>>word;
            int R= stoi(word);
            imgq[3*n*i+3*j]=R;
            infq>>word;
            int G= stoi(word);
            imgq[3*n*i+3*j+1]=G;
            infq>>word;
            int B= stoi(word);
            imgq[3*n*i+3*j+2]=B;
            //cout<<i<<","<<j<<endl;
            //cout<<R<<","<<G<<","<<B<<endl;
        }
        
        //cout<<i<<endl;
    }
    infq.close();
    cout<<"calculating avg"<<endl;
    float qavg;
    calcAvg(m,n,imgq,0,0,&qavg,n);
    cout<<"average: "<<qavg<<endl;
    cout<<"roatation started"<<endl;
    float* oimgq=new float[m*n*2];
    rotateimg(m,n,0.0,oimgq);
    float* pimgq=new float[m*n*2];
    rotateimg(m,n,45.0,oimgq);
    float* nimgq=new float[m*n*2];
    rotateimg(m,n,-45.0,oimgq);
    cout<<"rotation done"<<endl;
    int * imgc;
    int * imgcq;
    float* imgcqo;
    float* imgcqp;
    float* imgcqn;
    float *soln;
    //float* solnh=new float(M*N);
    hipMalloc (&imgc,(size_t) M*N*3*sizeof(int));
    hipMallocManaged(&soln,(size_t) 3*M*N*sizeof(float));
    hipMalloc (&imgcq,(size_t) m*n*3*sizeof(int));
    hipMalloc (&imgcqo,(size_t) m*n*3*sizeof(float));
    hipMalloc (&imgcqp,(size_t) m*n*3*sizeof(float));
    hipMalloc (&imgcqn,(size_t) m*n*3*sizeof(float));
    cout<<"memory allocated"<<endl;
    hipMemcpy(imgc,img,(size_t)sizeof(int)*M*N*3,hipMemcpyHostToDevice);
    hipMemcpy(imgcq,imgq,(size_t)sizeof(int)*m*n*3,hipMemcpyHostToDevice);
    hipMemcpy(imgcqo,oimgq,sizeof(float)*m*n*3,hipMemcpyHostToDevice);
    hipMemcpy(imgcqp,pimgq,sizeof(float)*m*n*3,hipMemcpyHostToDevice);
    hipMemcpy(imgcqo,nimgq,sizeof(float)*m*n*3,hipMemcpyHostToDevice);
    cout<<"memory copied"<<endl;
    cout<<"average: "<<qavg<<endl;
    query<<<((M*N)/1024 + 1), 1024>>>(M,N,m,n,qavg,imgcqo,imgcqp,imgcqn,imgc,imgcq,soln,th1,th2);
    cout<<"processing done"<<endl;
    //cudaMemcpy(solnh,soln,(size_t)sizeof(int)*M*N,cudaMemcpyDeviceToHost);
    hipDeviceSynchronize();
    float min = 100000.0;
    int idxi=-1;
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
            int idx=i*N+j;
            if (soln[3*(idx)]<=min && soln[3*(idx)+2]>0){
                min=soln[3*idx];
                idxi=idx;
                //cout<<i<<","<<j<<","<<soln[3*(i*N+j)]<<","<<soln[3*(i*N+j)+1]<<","<<soln[3*(i*N+j)+2]<<endl;
            }
        }
    }
    ofstream outf;
    outf.open(outfile.c_str(),ios::out);
    outf<<idxi/N<<" ";
    outf<<idxi%N<<" ";
    outf<<soln[idxi*3+2]-1<<" ";
    outf.close();
    hipFree(imgc);
    hipFree(soln);
    hipFree(imgcq);
    hipFree(imgcqo);
    hipFree(imgcqp);
    hipFree(imgcqn);

}